#include "hip/hip_runtime.h"
#include <hip/hip_math_constants.h>
extern "C" __global__

void minimum_distance(
    const float* rpos_ij_e,
    const float* tvecs_n,
    const long long int* batch_i,
    const long long int* edge_ij_e,
    const long long int E,
    const float* rveclens_n,
    const double cutoff_radius,
    float* dist2_min_e){
    
    const long long int tid = (long long int)blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= E) return;

    const long long int e = tid;
    const long long int i = edge_ij_e[e];
    const long long int n = batch_i[i];
    rpos_ij_e += e*3;
    const float r_ijx = rpos_ij_e[0];
    const float r_ijy = rpos_ij_e[1];
    const float r_ijz = rpos_ij_e[2];
    tvecs_n += n*9;
    const float t1_x = tvecs_n[0];
    const float t1_y = tvecs_n[1];
    const float t1_z = tvecs_n[2];
    const float t2_x = tvecs_n[3];
    const float t2_y = tvecs_n[4];
    const float t2_z = tvecs_n[5];
    const float t3_x = tvecs_n[6];
    const float t3_y = tvecs_n[7];
    const float t3_z = tvecs_n[8];
    
    rveclens_n += n*3;
    const float rvl1 = rveclens_n[0];
    const float rvl2 = rveclens_n[1];
    const float rvl3 = rveclens_n[2];
    
    float cutoff = (float)cutoff_radius;
    int R1 = LATTICE_RANGE, R2 = LATTICE_RANGE, R3 = LATTICE_RANGE;
    if (cutoff > 0.0f)
    {
        R1 = ceil((cutoff + 0.01f)*rvl1/(2.0*HIP_PI_F));
        R2 = ceil((cutoff + 0.01f)*rvl2/(2.0*HIP_PI_F));
        R3 = ceil((cutoff + 0.01f)*rvl3/(2.0*HIP_PI_F));

        #if MINIMUM_RANGE > 0
        R1 = max(R1, MINIMUM_RANGE);
        R2 = max(R2, MINIMUM_RANGE);
        R3 = max(R3, MINIMUM_RANGE);
        #endif
    }

    float d2min = 1e10;
    for (float n1 = -R1; n1 <= R1; n1++)
    for (float n2 = -R2; n2 <= R2; n2++)
    for (float n3 = -R3; n3 <= R3; n3++)
    {
        float dx = r_ijx + t1_x*n1 + t2_x*n2 + t3_x*n3;
        float dy = r_ijy + t1_y*n1 + t2_y*n2 + t3_y*n3;
        float dz = r_ijz + t1_z*n1 + t2_z*n2 + t3_z*n3;
        float d2 = dx*dx + dy*dy + dz*dz;
        d2min = fminf(d2min, d2);
    }
    dist2_min_e[e] = d2min;
}