#include "hip/hip_runtime.h"

extern "C" __global__
void fused_dpa_bwd_q_v3(
    const float* key_ihk,
    const float* gaij_eh,
    const long long int* edge_ij_e, 
    const long long int* e_start_i,
    const long long int N,
    const long long int H,
    const long long int E,
    float* gque_ihk
){
    const long long int tid = (long long int)blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= N*H) return;

    const long long int K = VPE_DIM;
    const long long int i = tid/H;
    const long long int h = tid%H;
    const long long int e_start = e_start_i[i];
    const long long int e_end = e_start_i[i+1];

    __shared__ float _gq[THREAD_NUM][K_HEAD_DIM+1];
    __shared__ float _rgq[THREAD_NUM][K_HEAD_DIM+1];
    float *gq = _gq[threadIdx.x];
    float *rgq = _rgq[threadIdx.x];

    #pragma unroll
    for (int k = 0; k < K_HEAD_DIM; k++){
        gq[k] = 0;
        rgq[k] = 0;
    }
    for (long long int e = e_start; e < e_end; e++)
    {
        long long int j = edge_ij_e[E+e];
        float g_softmax = gaij_eh[e*H+h];
        const float *key = key_ihk + (j*H+h)*K_HEAD_DIM;
        #pragma unroll
        for (int k = 0; k < K_HEAD_DIM; k++){
            rgq[k] += g_softmax*key[k];
        }
        
        if (((e-e_start) % RUNNING_SUM_LEN) == 0 || e == e_end-1){
            #pragma unroll
            for (int k = 0; k < K_HEAD_DIM; k++){
                gq[k] += rgq[k];
                rgq[k] = 0;
            }
        }
    }
    
    gque_ihk += (i*H+h)*K_HEAD_DIM;
    #pragma unroll
    for (int k = 0; k < K_HEAD_DIM; k++)
        gque_ihk[k] = gq[k];
}