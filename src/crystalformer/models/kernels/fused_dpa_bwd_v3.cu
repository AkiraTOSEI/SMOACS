#include "hip/hip_runtime.h"

extern "C" __global__
void fused_dpa_bwd_v3(
    const float* que_ihk,
    const float* key_ihk,
    const float* val_ihk,
    const float* taij_eh,
    const float* tbij_ehk,
    const long long int* batch_i,
    const long long int* edge_ij_e, 
    const long long int* e_start_i,
    const long long int N,
    const long long int H,
    const long long int E,
    const float* tprob_eh,
    const float* out_ihk,
    const float* gout_ihk,
    float* gque_ihk,
    float* gkey_ihk,
    float* gval_ihk,
    float* tgaij_eh,
    float* tgbij_ehk
){
    const long long int tid = (long long int)blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= N*H) return;

    const long long int K = VPE_DIM;
    const long long int j = tid/H;
    const long long int h = tid%H;
    const long long int n = batch_i[j];
    const long long int e_start = e_start_i[j];
    const long long int e_end = e_start_i[j+1];

    const float* que_k = &que_ihk[tid*K_HEAD_DIM];
    __shared__ float _v[THREAD_NUM][V_HEAD_DIM+1];
    __shared__ float _gv[THREAD_NUM][V_HEAD_DIM+1];
    __shared__ float _gk[THREAD_NUM][K_HEAD_DIM+1];
    __shared__ float _run_gv[THREAD_NUM][V_HEAD_DIM+1];
    __shared__ float _run_gk[THREAD_NUM][K_HEAD_DIM+1];
    float *v = _v[threadIdx.x];
    float *gv = _gv[threadIdx.x];
    float *gk = _gk[threadIdx.x];
    float *rgv = _run_gv[threadIdx.x];
    float *rgk = _run_gk[threadIdx.x];

    const float *v_src = val_ihk + (j*H+h)*V_HEAD_DIM;
    #pragma unroll
    for (int k = 0; k < V_HEAD_DIM; k++){
        gv[k] = 0;
        rgv[k] = 0;
        v[k] = v_src[k];
    }

    #pragma unroll
    for (int k = 0; k < K_HEAD_DIM; k++){
        gk[k] = 0;
        rgk[k] = 0;
    }

    if (tgbij_ehk != NULL && tbij_ehk != NULL ) {
        for (long long int e = e_start; e < e_end; e++)
        {
            long long int i = edge_ij_e[E+e];

            float pij = tprob_eh[e*H+h];
            const float *go = gout_ihk + (i*H+h)*V_HEAD_DIM;
            const float *o = out_ihk + (i*H+h)*V_HEAD_DIM;
            float *gb = tgbij_ehk + (e*H+h)*V_HEAD_DIM;
            const float *b = tbij_ehk + (e*H+h)*V_HEAD_DIM;
            float g_softmax = 0;
            #pragma unroll
            for (int k = 0; k < V_HEAD_DIM; k++){
                float t = go[k]*pij;
                rgv[k] += t;
                gb[k] = t;
                g_softmax += (v[k] + b[k] - o[k]) * t;
            }

            tgaij_eh[e*H+h] = g_softmax;

            const float *q = que_ihk + (i*H+h)*K_HEAD_DIM;
            #pragma unroll
            for (int k = 0; k < K_HEAD_DIM; k++){
                rgk[k] += g_softmax*q[k];
            }
            
            if (((e-e_start) % RUNNING_SUM_LEN) == 0 || e == e_end-1){
                #pragma unroll
                for (int k = 0; k < V_HEAD_DIM; k++){
                    gv[k] += rgv[k];
                    rgv[k] = 0;
                }
                #pragma unroll
                for (int k = 0; k < K_HEAD_DIM; k++){
                    gk[k] += rgk[k];
                    rgk[k] = 0;
                }
            }
        }
    } else {
        for (long long int e = e_start; e < e_end; e++)
        {
            long long int i = edge_ij_e[E+e];

            float pij = tprob_eh[e*H+h];
            const float *go = gout_ihk + (i*H+h)*V_HEAD_DIM;
            const float *o = out_ihk + (i*H+h)*V_HEAD_DIM;
            float g_softmax = 0;
            #pragma unroll
            for (int k = 0; k < V_HEAD_DIM; k++){
                float t = go[k]*pij;
                rgv[k] += t;
                g_softmax += (v[k] - o[k]) * t;
            }

            tgaij_eh[e*H+h] = g_softmax;

            const float *q = que_ihk + (i*H+h)*K_HEAD_DIM;
            #pragma unroll
            for (int k = 0; k < K_HEAD_DIM; k++){
                rgk[k] += g_softmax*q[k];
            }

            if (((e-e_start) % RUNNING_SUM_LEN) == 0 || e == e_end-1){
                #pragma unroll
                for (int k = 0; k < V_HEAD_DIM; k++){
                    gv[k] += rgv[k];
                    rgv[k] = 0;
                }
                #pragma unroll
                for (int k = 0; k < K_HEAD_DIM; k++){
                    gk[k] += rgk[k];
                    rgk[k] = 0;
                }
            }
        }
    }
    
    gval_ihk += (j*H+h)*V_HEAD_DIM;
    #pragma unroll
    for (int k = 0; k < V_HEAD_DIM; k++)
        gval_ihk[k] = gv[k];

    gkey_ihk += (j*H+h)*K_HEAD_DIM;
    #pragma unroll
    for (int k = 0; k < K_HEAD_DIM; k++)
        gkey_ihk[k] = gk[k];
}