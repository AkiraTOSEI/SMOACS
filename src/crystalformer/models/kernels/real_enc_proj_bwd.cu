#include "hip/hip_runtime.h"
#include <hip/hip_math_constants.h>
extern "C" __global__

void real_enc_proj_bwd(
    const float* a_ik,
    const float* rpos_ij_e,
    //const float* dist2_min_e,
    const float* tvecs_n,
    const long long int* batch_i,
    const long long int* edge_ij_e,
    const long long int* e_start_i,
    const float* z_ek,
    const float* gz_ek,
    const float* gv_ekd,
    const long long int N,
    const long long int H,
    const long long int E,
    //const long long int K_,
    const double dist_max,
    const double wscale,
    const float* W_k,
    const long long int W_num,
    const float* rveclens_n,
    const double cutoff_radius,
    float* ga_ik,
    float* gW_k){
    
    const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= N*H) return;

    const unsigned int k = tid%H;
    const unsigned int i = tid/H;
    const unsigned int n = batch_i[i];
    tvecs_n += n*9;
    const float t1_x = tvecs_n[0];
    const float t1_y = tvecs_n[1];
    const float t1_z = tvecs_n[2];
    const float t2_x = tvecs_n[3];
    const float t2_y = tvecs_n[4];
    const float t2_z = tvecs_n[5];
    const float t3_x = tvecs_n[6];
    const float t3_y = tvecs_n[7];
    const float t3_z = tvecs_n[8];
    const float a = a_ik[i*H + k];
    const unsigned int e_end = e_start_i[i+1];
    #if VPE_DIM > 0
    __shared__ float shared_gv[THREAD_NUM][VPE_DIM+1];
    __shared__ float shared_v[THREAD_NUM][VPE_DIM+1];
    float *sv = shared_v[threadIdx.x];
    float *gW = NULL;
    if (gW_k != NULL && (W_num == N || W_num == 1)){
        gW = &gW_k[(i*H+k)*V_HEAD_DIM*VPE_DIM];
        for (int dim = 0; dim < V_HEAD_DIM*VPE_DIM; dim++)
            gW[dim] = 0;
    }
    #endif

    rveclens_n += n*3;
    const float rvl1 = rveclens_n[0];
    const float rvl2 = rveclens_n[1];
    const float rvl3 = rveclens_n[2];
    
    float cutoff = (float)cutoff_radius;
    int R1 = LATTICE_RANGE, R2 = LATTICE_RANGE, R3 = LATTICE_RANGE;
    if (cutoff != 0.0f)
    {
        if (cutoff < 0) {
            // Better sync the threads in each block?
            // -> disabled due to thread stucking
            // float a_max = a;
            // for (int t = 0; t < THREAD_NUM; t++)
            //     a_max = max(a_max, a_ik[i*H + t]);
            //cutoff = sqrt(-0.5f/a_max)*(-cutoff);
            cutoff = sqrt(-0.5f/a)*(-cutoff);
        }
        R1 = ceil((cutoff + 0.01f)*rvl1/(2.0*HIP_PI_F));
        R2 = ceil((cutoff + 0.01f)*rvl2/(2.0*HIP_PI_F));
        R3 = ceil((cutoff + 0.01f)*rvl3/(2.0*HIP_PI_F));
        float cutoff2 = cutoff*cutoff;

        #if MINIMUM_RANGE > 0
        R1 = max(R1, MINIMUM_RANGE);
        R2 = max(R2, MINIMUM_RANGE);
        R3 = max(R3, MINIMUM_RANGE);
        #endif
    }
    
    float sum = 0;
    float sum_v = 0;
    for (unsigned int e = e_start_i[i]; e < e_end; e++)
    {
        const unsigned int j = edge_ij_e[E+e];
        const float r_ijx = rpos_ij_e[e*3+0];
        const float r_ijy = rpos_ij_e[e*3+1];
        const float r_ijz = rpos_ij_e[e*3+2];
        const unsigned int ek = e*H+k;
        const float z = z_ek[ek];
        const float gz = gz_ek[ek];

        #if VPE_DIM > 0
        float *sgv = shared_gv[threadIdx.x];
        if (gW_k == NULL){
            const float *gv = &gv_ekd[ek*VPE_DIM];
            #pragma unroll
            for (int dim = 0; dim < VPE_DIM; dim++) {
                sgv[dim] = gv[dim];
            }
        } else {
            // Compute backward of v' = Wv, as gW = (gv')^T * v
            const float *gv = &gv_ekd[ek*V_HEAD_DIM];
            unsigned int w_ind = 0;
            if (W_num == 1){
                w_ind = 0;
            } else if (W_num == E) {
                w_ind = e;
            } else if (W_num == N) {
                w_ind = i;
            }
            const float *W = &W_k[(w_ind*H+k)*V_HEAD_DIM*VPE_DIM];
            #pragma unroll
            for (int dim = 0; dim < VPE_DIM; dim++)
                sgv[dim] = 0;
            #pragma unroll
            for (int wdim = 0; wdim < V_HEAD_DIM; wdim++){
                float gv_val = gv[wdim];
                #pragma unroll
                for (int dim = 0; dim < VPE_DIM; dim++){
                    sgv[dim] += W[wdim*VPE_DIM+dim]*gv_val;
                    //sgv[dim] += (*W++)*gv_val;
                }
            }

            // for gW
            if (W_num == E){
                gW = &gW_k[(e*H+k)*V_HEAD_DIM*VPE_DIM];
                for (int dim = 0; dim < V_HEAD_DIM*VPE_DIM; dim++)
                    gW[dim] = 0;
            }
        }
        #endif

        float px_avr = 0;
        float pbg_avr = 0;
        const float reci_ws_sqrt2 = 1.0f/((float)wscale*sqrt(2.0f));
        const float mu0 = (float)dist_max/VPE_DIM;
        #if VPE_DIM > 0
        if (gW_k != NULL){
            #pragma unroll
            for (int dim = 0; dim < VPE_DIM; dim++)
                sv[dim] = 0;
        }
        #endif
        for (float n1 = -R1; n1 <= R1; n1++)
        for (float n2 = -R2; n2 <= R2; n2++)
        for (float n3 = -R3; n3 <= R3; n3++)
        {
            float dx = r_ijx + t1_x*n1 + t2_x*n2 + t3_x*n3;
            float dy = r_ijy + t1_y*n1 + t2_y*n2 + t3_y*n3;
            float dz = r_ijz + t1_z*n1 + t2_z*n2 + t3_z*n3;
            float d2 = dx*dx + dy*dy + dz*dz;
            // float dx = fmaf(t1_x, n1, fmaf(t2_x, n2, fmaf(t3_x, n3, r_ijx)));
            // float dy = fmaf(t1_y, n1, fmaf(t2_y, n2, fmaf(t3_y, n3, r_ijy)));
            // float dz = fmaf(t1_z, n1, fmaf(t2_z, n2, fmaf(t3_z, n3, r_ijz)));
            // float d2 = fmaf(dx,dx, fmaf(dy,dy, dz*dz));
            float p = expf(a*d2 - z);
            float px = d2*p;
            px_avr += px;

            #if VPE_DIM > 0
            float bg = 0;
            float b = -sqrtf(d2)/mu0*reci_ws_sqrt2;
            #pragma unroll
            for (int dim = 0; dim < VPE_DIM; dim++)
            {
                b += reci_ws_sqrt2;
                float gauss = expf(-b*b);
                bg += gauss*sgv[dim];
                sv[dim] += gauss*p;
            }
            sum_v += px*bg;
            pbg_avr += p*bg;
            #endif
        }
        /*
        b: (E, 1, R, K)
        x: (E, 1, R, 1)
        y: (N, H, 1, 1)
        z: (E, H, 1, K)
        g: (E, H, 1, K)
        p: (E, H, R, 1)

        (E,H,R,K)   (E,H,R,1)     (E,H,R,K)       (E,H,1,K): (E,H,R,1)*(E,1,R,K)*(E,H,1,K)
        dz/dye    =    p*x    * (    b*g     -    (p*b*g).sum(axis=R))

        (E,H,1,1)
        dz/dyi    = (dz/dye).sum(axis=R,K).sum_for_j()

                     (E,H,R,1)*(E,H,R,1)                (E,H,1,1)        *(E,H,1,1)
        dz/dye    =    (p*x)  *(b*g).sum(axis=K)    -   (p*x).sum(axis=R)*(p*b*g).sum(axis=R,K))
        */

        sum += px_avr*gz;
        sum_v -= px_avr*pbg_avr;
        
        #if VPE_DIM > 0
        if (gW_k != NULL){
            const float *gv = &gv_ekd[ek*V_HEAD_DIM];
            #pragma unroll
            for (int wdim = 0; wdim < V_HEAD_DIM; wdim++){
                float gv_val = gv[wdim];
                #pragma unroll
                for (int dim = 0; dim < VPE_DIM; dim++){
                    //*(_sgw++) += sv[dim]*gv_val;
                    gW[wdim*VPE_DIM+dim] += sv[dim]*gv_val;
                }
            }
        }
        #endif
    }
    ga_ik[tid] = sum + sum_v;
}