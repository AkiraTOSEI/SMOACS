#include "hip/hip_runtime.h"
#include "models/kernels/pairwise_sum.cuh"

extern "C" __global__
void fused_dpa_fwd(
    const float* que_ihk,
    const float* key_ihk,
    const float* val_ihk,
    const float* aij_eh,
    const float* bij_ehk,
    const long long int* edge_ij_e, 
    const long long int* e_start_i,
    const long long int N,
    const long long int H,
    const long long int E,
    float* prob_eh,
    float* out_ihk
){
    const long long int tid = (long long int)blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= N*H) return;
    const long long int i = tid/H;
    const long long int h = tid%H;
    const long long int e_start = e_start_i[i];
    const long long int e_end = e_start_i[i+1];

    #if 0
    // Read q from global mem.
    const float* que_k = &que_ihk[tid*K_HEAD_DIM];
    #else
    // Load q onto shared mem.
    __shared__ float _que_k[THREAD_NUM][K_HEAD_DIM+1];
    que_ihk += tid*K_HEAD_DIM;
    float* que_k = _que_k[threadIdx.x];
    #pragma unroll
    for (int k = 0; k < K_HEAD_DIM; k++){
        que_k[k] = que_ihk[k];
    }
    #endif
    
    __shared__ float _attns[THREAD_NUM][MAX_SYSTEM_SIZE+1];
    __shared__ float _output[THREAD_NUM][V_HEAD_DIM+1];
    float *attns = _attns[threadIdx.x];
    float *output = _output[threadIdx.x];

    float max_attn = -1e20;
    int e_count = 0;
    for (long long int e = e_start; e < e_end; e++)
    {
        long long int j = edge_ij_e[E+e];
        const float* key_k = &key_ihk[(j*H+h)*K_HEAD_DIM];

        float attn = 0;
        #pragma unroll
        for (int k = 0; k < K_HEAD_DIM; k++){
            attn += que_k[k]*key_k[k];
        }
        if (aij_eh != NULL)
           attn += aij_eh[e*H+h];
        max_attn = max(max_attn, attn);
        attns[e_count] = attn;
        e_count++;
    }
    
    float sum = 0;
    for (int j = 0; j < e_count; j++)
    {
        float v = exp(attns[j] - max_attn);
        attns[j] = v;
        sum += v;
    }

    // Compute pairwise sum for better numerical accuracy.
    constexpr int BS = 64;
    for (int j = e_count; j < (e_count+BS-1)/BS*BS; j++)
        attns[j] = 0.0f;
    float s[MAX_SYSTEM_SIZE_POW2/BS] = {0.0f};
    for (int j = 0; j < (e_count+BS-1)/BS; j++){
        s[j] = fixed_length_pairwise_sum<BS>(&attns[j*BS]);
    }
    sum = fixed_length_pairwise_sum<MAX_SYSTEM_SIZE_POW2/BS>(s);


    for (int j = 0; j < e_count; j++){
        attns[j] /= sum;
        prob_eh[(e_start+j)*H+h] = attns[j];
    }

    #pragma unroll
    for (int k = 0; k < V_HEAD_DIM; k++){
        output[k] = 0;
    }

    e_count = 0;
    if (bij_ehk != NULL) {
        for (long long int e = e_start; e < e_end; e++)
        {
            long long int j = edge_ij_e[E+e];
            const float* val_k = &val_ihk[(j*H+h)*V_HEAD_DIM];
            const float* bij_k = &bij_ehk[(e*H+h)*V_HEAD_DIM];

            float attn = attns[e_count];
            #pragma unroll
            for (int k = 0; k < V_HEAD_DIM; k++){
                output[k] += (val_k[k]+bij_k[k])*attn;
            }
            e_count++;
        }
    } else {
        for (long long int e = e_start; e < e_end; e++)
        {
            long long int j = edge_ij_e[E+e];
            const float* val_k = &val_ihk[(j*H+h)*V_HEAD_DIM];

            float attn = attns[e_count];
            #pragma unroll
            for (int k = 0; k < V_HEAD_DIM; k++){
                output[k] += val_k[k]*attn;
            }
            e_count++;
        }
    }

    out_ihk += tid*V_HEAD_DIM;
    #pragma unroll
    for (int k = 0; k < V_HEAD_DIM; k++)
        out_ihk[k] = output[k];
}